#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<>
#include<iostream>

__global__ void matixMultiplication(int *gA, int*gB, int *gC, int row, int col) {
	unsigned int idx = blockDim.x*blockIdx.x + threadIdx.x;
	unsigned int idy = blockDim.y*blockIdx.y + threadIdx.y;
	if (idx < row && idy < col){
		int cvalue = 0;
		for (int i = 0; i < col; i++){
			cvalue+=gA[idy*row+i] * gB[idx+(col*i)];
			//printf("The C index is %d and the A index is %d B is %d\n", idy*row + idx, idy*row + i, idx + (col*i));			
			//__syncthreads();
		}
		gC[idy*row + idx] = cvalue;
		//__syncthreads();
	}
}
int main()
{
	int row = 800;
	int col = 800;
	int *A,*B,*C, *devA, *devB, *devC,*hostC;

	// Allocating memory for host matrices A and B
	A = new int[row*col];
	B = new int[row*col];

	//allocating memory to store the output matrix C
	C = new int[row*col];
	hostC = new int[row*col];
	// initialising values for Matrix A
	for (int i = 0; i < row*col; i++) A[i] = i;
	// initialising values for Matrix B
	for (int i = 0; i < row*col; i++) B[i] = i;

	// Allocating memory for host matrices devA, devB and output matrix devC
	hipMalloc((void**)&devA, row*col*sizeof(int));
	hipMalloc((void**)&devB, row*col * sizeof(int));
	hipMalloc((void**)&devC, row*col * sizeof(int));

	// copying the data from hos tot device
	

	//host multiplication
	for (int j = 0; j < col; j++)
	{
		
		for (int i = 0; i < row; i++)
		{
			int temp = 0;
			for (int k = 0; k < col; k++)
			{
				temp += A[col*j + k] * B[row*k + i];
				//std::cout << col * j + k << " " << row * k + i << std::endl;
			}
			//std::cout << std::endl << col * j + i << std::endl;
			C[col*j + i] = temp;
		}

	}
	//for (int i = 0; i < row*col; i++)std::cout << C[i] << " " << i << std::endl;
	//Threads per block is 32
	hipMemcpy(devA, A, row*col * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(devB, B, row*col * sizeof(int), hipMemcpyHostToDevice);
	dim3 block(32,32);
	dim3 grid((row + block.x - 1) / block.x, (col + block.y - 1) / block.y);

	matixMultiplication << <block, grid >> > (devA, devB,devC,row,col);

	hipMemcpy(hostC, devC, row*col * sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i < row*col; i++)
	{
		if (hostC[i] != C[i])
		{
			std::cout << hostC[i] << " " << C[i] << std::endl;
			std::cout << " You got some work to do chap !!" << std::endl;
			goto exit;
		}
	}
	std::cout << "You made it !!" << std::endl;
exit:
	delete(A);
	delete(B);
	delete(C);
	delete(hostC);
	hipFree(devA);
	hipFree(devB);
	hipFree(devC);
	return 0;
}